/*
rkrish11 Rahul Krishna
*/
#include "hip/hip_runtime.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#define SEED 35791246



__global__ void init_stuff(hiprandState *state, int count) {
// This sets a random number seed for all the threads
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 if (idx<count)
  hiprand_init(1337, idx, 0, &state[idx]);
}


__global__ void cudaMonte(double* pi, int count, hiprandState* state) {

  // Perfome MC simulation on the threads
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  double x,y,z;

  if (id<count) {
    x = (double)hiprand_uniform(&state[id]);
    y = (double)hiprand_uniform(&state[id]);
    z = x*x+y*y;
    if (z<=1) pi[id]=1;
    else pi[id]=0;
  }
  __syncthreads();

  // Find the total number of points that lie inside the quadrant of the cirle
  for (int i=1; i<count;i++) {
    pi[0]+=pi[i];
  }
}

int main(int argc, char** argv) {
  int niter=0;
  double pi;
  double* d_pi;
  hiprandState *d_state;

  printf("Enter the number of iterations used to estimate pi: ");
  scanf("%d",&niter);

  double* h_pi = new double[niter];

  if (hipMalloc(&d_pi, sizeof(int)*niter) != hipSuccess) {
      printf("Error in memory allocation.\n");
      return 0;
  }
  if (hipMalloc(&d_state, sizeof(hiprandState)*niter) != hipSuccess) {
      printf("Error in memory allocation for random state.\n");
      return 0;
  }
  if (hipMemcpy (d_pi, h_pi, sizeof(int)*niter, hipMemcpyHostToDevice) != hipSuccess) {
      printf("Error in copy from host to device.\n");
      hipFree(d_pi);
      return 0;
  }

  // Number of threads = 1024, number of blocks = (int) (niter/threads)+1
  init_stuff<<<(int) niter/1024+1, 1024>>>(d_state, niter);
  cudaMonte<<<(int) niter/1024+1, 1024>>>(d_pi, niter, d_state);

  if (hipMemcpy (h_pi, d_pi, sizeof(int)*niter, hipMemcpyDeviceToHost) != hipSuccess) {
      printf("Error in copy from device to host.\n");
      delete[] h_pi;
      hipFree(d_pi);
      return 0;
  }

  // Final Estimate of pi
  pi= (double) h_pi[0]/niter*4;
  printf("# of trials= %d , estimate of pi is %g \n",niter,pi);

}
