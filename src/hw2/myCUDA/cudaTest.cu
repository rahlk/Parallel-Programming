#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void cudaADD(int* a, int* b) {
  a[0]+=b[0];
}

int main(){
  int a=5, b=6;
  int *c_a, *c_b;

  // Allocate memory for CUDA
  hipMalloc(&c_b, sizeof(int));
  hipMalloc(&c_a, sizeof(int));

  // Transfer data to GPU from CPU
  hipMemcpy(c_a, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c_b, &b, sizeof(int), hipMemcpyHostToDevice);

  // Run on GPU
  cudaADD<<<1,1>>>(c_a, c_b);

  // Transfer for GPU to CPU
  hipMemcpy(&a, c_a, sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", a);

  // Free allocated memory
  hipFree(c_a);
  hipFree(c_b);

  return 0;
}
