#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <mpi.h>

#define __DEBUG
#define VSQR 0.1
#define TSCALE 1.0
#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)


extern int tpdt(double *t, double dt, double end_time);

/**************************************
* void __cudaSafeCall(hipError_t err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}

__device__ double f_CUDA(double p, double t)
{
  return -__expf(-TSCALE * t) * p;
}

__global__ void evolve9ptCUDA(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t) {
  int idx = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
  int i = idx / n;
  int j = idx % n;
  if(!(i == 0 || i == n - 1 || j == 0 || j == n - 1))
    un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
  else un[idx] = 0.;
}
__global__ void evolve9ptCUDA_mpi(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t, int startRow) {
  int idx = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
  int i = idx / n;
  int j = idx % n;
  idx = idx + startRow*n;
  if(!(i == 0 || i == n - 1 || j == 0 || j == n - 1))
    un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
  else un[idx] = 0.;
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads)
{
	hipEvent_t kstart, kstop;
	float ktime;
        
	double *un, *uc, *uo, *pb, *temp;
  double t, dt;
        /* Set up device timers */  
	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipEventCreate(&kstart));
	CUDA_CALL(hipEventCreate(&kstop));

  t = 0.;
  dt = h/2.;

  hipMalloc((void **)&un, sizeof(double) * n * n);
  hipMalloc((void **)&uc, sizeof(double) * n * n);
  hipMalloc((void **)&uo, sizeof(double) * n * n);
  hipMalloc((void **)&pb, sizeof(double) * n * n);

  hipMemcpy(uo, u0, sizeof(double) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(uc, u1, sizeof(double) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(pb, pebbles, sizeof(double) * n * n, hipMemcpyHostToDevice);

  dim3 block_dim(nthreads, nthreads,1);
  dim3 grid_dim(n/nthreads, n/nthreads,1);

	/* Start GPU computation timer */
	CUDA_CALL(hipEventRecord(kstart, 0));
  while(1)
  {
    evolve9ptCUDA<<<grid_dim, block_dim>>>(un, uc, uo, pb, n, h, dt, t);
    temp = uc;
    uc = un;
    un = uo;
    uo = temp;
    if(!tpdt(&t, dt, end_time))
      break;
  }
	hipMemcpy(u, uc, sizeof(double) * n * n, hipMemcpyDeviceToHost);
        /* Stop GPU computation timer */
	CUDA_CALL(hipEventRecord(kstop, 0));
	CUDA_CALL(hipEventSynchronize(kstop));
	CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
	printf("GPU computation: %f msec\n", ktime);

	hipFree(un);
  hipFree(uc);
  hipFree(uo);
  hipFree(pb);

	/* timer cleanup */
	CUDA_CALL(hipEventDestroy(kstart));
	CUDA_CALL(hipEventDestroy(kstop));
}
void run_gpu_mpi(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads, int numproc, int rank)
{
        
  int startRow, endRow, rowsPerPart, totalElements;
  rowsPerPart = n/numproc;
  startRow = rowsPerPart * rank;
  endRow = rowsPerPart * (rank + 1);
  double *un, *uc, *uo, *pb, *temp;
  double t, dt;

  t = 0.;
  dt = h/2.;

  double *uc_host, *uo_host, *un_host;
  uc_host=(double *)malloc(sizeof(double)*n*n);
  uo_host=(double *)malloc(sizeof(double)*n*n);
  un_host=(double *)malloc(sizeof(double)*n*n);
  memset(un_host, 0, sizeof(double)*n*n);

  hipMalloc((void **)&un, sizeof(double) * n * n);
  hipMalloc((void **)&uc, sizeof(double) * n * n);
  hipMalloc((void **)&uo, sizeof(double) * n * n);
  hipMalloc((void **)&pb, sizeof(double) * n * n);

  hipMemcpy(uo, u0, sizeof(double) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(uc, u1, sizeof(double) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(un, un_host, sizeof(double) * n * n, hipMemcpyHostToDevice);
  hipMemcpy(pb, pebbles, sizeof(double) * n * n, hipMemcpyHostToDevice);

  dim3 block_dim(nthreads, nthreads,1);
  dim3 grid_dim(n/(nthreads * 2), n/(nthreads * 2),1);

    int iter = 0;
  while(1)
  {
    MPI_Request *requestList;
    int reqCount = 0;
    requestList = (MPI_Request *)malloc(sizeof(MPI_Request)*8);
    evolve9ptCUDA_mpi<<<grid_dim, block_dim>>>(un, uc, uo, pb, n, h, dt, t, startRow);
    if(!tpdt(&t, dt, end_time))
      break;
    printf("%d iteration in rank %d with startRow as %d.\n", iter, rank, startRow);
    iter++;
    hipMemcpy(uc_host, uc, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(un_host, un, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    printf("Copied from device.\n");
    if(rank!=0)
    {
      MPI_Irecv(un_host + (rowsPerPart * rank - 1)*n , n, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, &(requestList[reqCount]));
      MPI_Irecv(uc_host + (rowsPerPart * rank - 1)*n , n, MPI_DOUBLE, rank - 1, 1, MPI_COMM_WORLD, &(requestList[reqCount+1]));
      reqCount = reqCount + 2;
    }
    if(rank != numproc - 1)
    {
      MPI_Irecv(un_host + rowsPerPart * (rank + 1) * n, n, MPI_DOUBLE, rank + 1, 0, MPI_COMM_WORLD, &requestList[reqCount]);
      MPI_Irecv(uc_host + rowsPerPart * (rank + 1) * n, n, MPI_DOUBLE, rank + 1, 1, MPI_COMM_WORLD, &requestList[reqCount+1]);
      reqCount = reqCount + 2;
    }
    if(rank != 0)
    { 
      MPI_Isend(un_host + (rowsPerPart * rank)*n, n, MPI_DOUBLE, rank - 1, 0, MPI_COMM_WORLD, &requestList[reqCount]);
      MPI_Isend(uc_host + (rowsPerPart * rank)*n, n, MPI_DOUBLE, rank - 1, 1, MPI_COMM_WORLD, &requestList[reqCount+1]);
      reqCount = reqCount + 2;
    }
    if(rank != numproc-1)
    {
      MPI_Isend(un_host + (rowsPerPart * (rank + 1) - 1)*n, n, MPI_DOUBLE, rank+1, 0, MPI_COMM_WORLD, &requestList[reqCount]);
      MPI_Isend(uc_host + (rowsPerPart * (rank + 1) - 1)*n, n, MPI_DOUBLE, rank+1, 1, MPI_COMM_WORLD, &requestList[reqCount+1]);
      reqCount = reqCount + 2;
    }
    MPI_Waitall(reqCount, requestList, MPI_STATUS_IGNORE);
    hipMemcpy(uo, uc_host, sizeof(double)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(uc, un_host, sizeof(double)*n*n, hipMemcpyHostToDevice);
  }
  hipMemcpy(u, un, sizeof(double) * n * n, hipMemcpyDeviceToHost);

  hipFree(un);
  hipFree(uc);
  hipFree(uo);
  hipFree(pb);
}
