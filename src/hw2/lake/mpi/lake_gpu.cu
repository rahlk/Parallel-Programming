#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "mpi.h"
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define __DEBUG
#define VSQR 0.1
#define TSCALE 1.0
#define XMIN 0.0
#define XMAX 1.0
#define YMIN 0.0
#define YMAX 1.0
#define USE_MATH_DEFINES
#define MAX_PSZ 10
#define TSCALE 1.0
#define VSQR 0.1
#define NINEPTSTENCIL 1
#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

void init(double *u, double *pebbles, int n);
void evolve(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
void evolve9pt(double *un, double *uc, double *uo, double *pebbles, int n, int rank, double *row, double *col, double *indv, double h, double dt, double t);
void evolve9pt_1(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t);
int tpdt(double *t, double dt, double end_time);
void print_heatmap(char *filename, double *u, int n, double h);
void init_pebbles(double *p, int pn, int n);
void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time);
void transfer(double *from, double *to, int r, int n, bool dir);
void dest(double *source, double *row, double *col, double *indv, int *hor, int *ver, int *diag, int rank, int size);
extern void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads);


/**************************************
* void __cudaSafeCall(hipError_t err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}

__device__ double f_CUDA(double p, double t)
{
  return -__expf(-TSCALE * t) * p;
}

__global__ void evolve9ptCUDA(double *un, double *uc, double *uo, double *pebbles, int n, int rank, double *row, double *col, double *indv, double h, double dt, double t) {
  int idx = (blockIdx.x * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y + threadIdx.x * blockDim.x + threadIdx.y;
  int i = idx / n;
  int j = idx % n;
  if(!(i == 0 || i == n - 1 || j == 0 || j == n - 1)) {
    un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
  }
  else {
    // Code for the fringe regions goes here...
    switch (rank) {
      case 1:
        if (i==0 || j==0) {
          un[idx]=0.;
        }
        else if(i==n-1 && j==n-1) { // Bottom right corner
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + col[i] + row[j] + uc[idx - n] + 0.25*(row[j-1] + *indv + uc[idx - n - 1] + col[i-1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(j==n-1 && i<n-1) { // Right edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + col[i] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + col[i + 1] + uc[idx - n - 1] + col[i-1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(i==n-1 && j<n-1) { // Bottom edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + row[j] + uc[idx - n] + 0.25*(row[j - 1] + row[j + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        break;

      case 2:
        if (i==0 || j==n-1){
          un[idx]=0.;
        }
        else if(i==n-1 && j==0) { // Bottom left corner
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((col[i] + uc[idx+1] + row[j] + uc[idx - n] + 0.25*(*indv + row[j + 1] + col[i - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(j==0 && i<n-1) { // Left edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((col[i] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(col[i+1] + uc[idx + n + 1] + col[i-1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(i==n-1 && j>0) { // Bottom Edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + row[j] + uc[idx - n] + 0.25*(row[j - 1] + row[j + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        break;

      case 3:
        if (i==n-1 || j==0){
          un[idx]=0.;
        }
        else if(i==0 && j==n-1) { // Top right corner
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + col[i] + uc[idx + n] + row[j] + 0.25*(uc[idx + n - 1] + col[i+1] + row[j-1] + *indv)- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(j==n-1 && i>0) { // Left edge
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + col[i] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + col[i+1] + uc[idx - n - 1] + col[i-1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(i==0 && j<n-1) { // Top Edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + row[j] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + row[j-1] + row[j+ 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        break;

      case 4:
        if (i==n-1 || j==n-1){
          un[idx]=0.;
        }
        else if(i==0 && j==0) { // Top left corner
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((col[i] + uc[idx+1] + uc[idx + n] + row[j] + 0.25*(col[i+1] + uc[idx+n+1] + *indv + row[j+1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(j==0 && i>0) { // Right edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((col[i] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(col[i+1] + uc[idx + n + 1] + col[i-1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        else if(i==0 && j>0) { // Top Edge
          un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + row[j] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + row[j-1] + row[j+1])- 5 * uc[idx])/(h * h) + f_CUDA(pebbles[idx],t));
        }
        break;
    }
  }}


int main(int argc, char *argv[])
{

  int   numproc, rank;

  MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Status *status;
  MPI_Request *request;
  request = (MPI_Request *) malloc(numproc * sizeof(MPI_Request));
  status = (MPI_Status *) malloc(numproc * sizeof(MPI_Status));

  int     npoints   = 256;// atoi(argv[1]);
  int     npebs     = 3;// atoi(argv[2]);
  double  end_time  = 1.00;// (double)atof(argv[3]);
  int     nthreads  = 1024;// atoi(argv[4]);
  int 	  narea	    = npoints * npoints;
  bool    once      = true;
  int size=(npoints/2)*(npoints/2);

  double t, dt;
  double h = (XMAX - XMIN)/npoints;

  if (rank == 0) {

    double *u_i0, *u_i1;
    double *u_cpu, *u_gpu, *pebs;
    double *peb, *n_cpu; //1, *n_cpu2, *n_cpu3, *n_cpu4;
    double elapsed_cpu, elapsed_gpu;
    struct timeval cpu_start, cpu_end, gpu_start, gpu_end;

    peb   = (double*)malloc(sizeof(double) * size);
    u_i0  = (double*)malloc(sizeof(double) * narea);
    u_i1  = (double*)malloc(sizeof(double) * narea);
    u_cpu = (double*)malloc(sizeof(double) * narea);
    u_gpu = (double*)malloc(sizeof(double) * narea);
    n_cpu = (double*)malloc(sizeof(double) * size);

    pebs = (double*)malloc(sizeof(double) * narea);
    printf("Rank0: Running a (%d x %d) grid, until %f, with %d threads\n", npoints, npoints, end_time, nthreads);
    init_pebbles(pebs, npebs, npoints);
    init(u_i0, pebs, npoints);
    init(u_i1, pebs, npoints);

    // Initial
    run_cpu(u_gpu, u_i0, u_i1, pebs, npoints, h, end_time);
    print_heatmap("lake_cpu_f.dat", u_gpu, npoints, h);

    // Tranfer to MPI nodes
    int i;
    for (i=1; i<numproc; i++) {
      transfer(pebs, peb, i, npoints, true); //get corresponding data
      MPI_Send(peb,size, MPI_DOUBLE, i, i, MPI_COMM_WORLD);
      }

      /*-----------------------------------*/
      /* Stitch individual nodes together */
      /*
      MPI_Recv(n_cpu, size, MPI_DOUBLE, 1, 11, MPI_COMM_WORLD, &status[1]);
      transfer(n_cpu, u_cpu, 1, npoints, false);

      MPI_Recv(n_cpu, size, MPI_DOUBLE, 2, 12, MPI_COMM_WORLD, &status[2]);
      transfer(n_cpu, u_cpu, 2, npoints, false);

      MPI_Recv(n_cpu, size, MPI_DOUBLE, 3, 13, MPI_COMM_WORLD, &status[3]);
      transfer(n_cpu, u_cpu, 3, npoints, false);

      MPI_Recv(n_cpu, size, MPI_DOUBLE, 4, 14, MPI_COMM_WORLD, &status[4]);
      transfer(n_cpu, u_cpu, 4, npoints, false);

      // Save final Image
      print_heatmap("lake_cpu_mpi.dat", u_cpu, npoints, h);
      */
      /*-----------------------------------*/


    }

  else {
      /* For Reference:

      + : Fringe edges
      X : Diagonal fringe point

      |``````````````````+|+````````````````|
      |                  +|+                |
      |                  +|+                |
      |     Rank 1       +|+     Rank 2     |
      |                  +|+                |
      |++++++++++++++++++X|X++++++++++++++++|
      |++++++++++++++++++X|X++++++++++++++++|
      |                  +|+                |
      |                  +|+                |
      |     Rank 3       +|+     Rank 4     |
      |                  +|+                |
      |                  +|+                |
      |                  +|+                |
      ```````````````````````````````````````
      */

    hipEvent_t kstart, kstop;
    float ktime;

    int number_amount;
    double *un  , *u0  , *u1  , *uc      , *uo, *pebble;
    double *d_un, *d_uc, *d_uo, *d_pebble, *d_temp;
    int n = npoints/2;

    u0     = (double*)malloc(sizeof(double) * n*n);
    u1     = (double*)malloc(sizeof(double) * n*n);
    un     = (double*)malloc(sizeof(double) * n*n);
    uc     = (double*)malloc(sizeof(double) * n*n);
    uo     = (double*)malloc(sizeof(double) * n*n);
    pebble = (double*)malloc(sizeof(double) * n*n);

    MPI_Recv(pebble, size, MPI_DOUBLE, 0, rank, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

    /*-----------------------------------*/
    /* Sanity Check!*/ /*
    MPI_Get_count(&status, MPI_INT, &number_amount);
    printf("1 received %d numbers from 0. Message source = %d, "
           "tag = %d\n",
           number_amount, status.MPI_SOURCE, status.MPI_TAG);
    /*-----------------------------------*/

    init(u0, pebble, npoints/2);
    init(u1, pebble, npoints/2);

    // Begin Timer
    t = 0.;
    dt = h / 2.;

    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipEventCreate(&kstart));
    CUDA_CALL(hipEventCreate(&kstop));

    hipMalloc((void **)&d_un, sizeof(double) * n * n);
    hipMalloc((void **)&d_uc, sizeof(double) * n * n);
    hipMalloc((void **)&d_uo, sizeof(double) * n * n);
    hipMalloc((void **)&d_pebble, sizeof(double) * n * n);

    hipMemcpy(d_uo, u0,    sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_uc, u1,    sizeof(double) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(d_pebble, pebble, sizeof(double) * n * n, hipMemcpyHostToDevice);

    dim3 block_dim(nthreads, nthreads,1);
    dim3 grid_dim(n/nthreads, n/nthreads,1);

    /* Start GPU computation timer */
    CUDA_CALL(hipEventRecord(kstart, 0));

    while(1) {
      // What to send where
      double *row, *col, *indv;
      row = (double*)malloc(sizeof(double) * npoints/2);
      col = (double*)malloc(sizeof(double) * npoints/2);
      indv = (double*)malloc(sizeof(double));

      int *hor, *ver, *diag;

      hor  = (int*)malloc(sizeof(int));
      ver  = (int*)malloc(sizeof(int));
      diag = (int*)malloc(sizeof(int));

      dest(un, row, col, indv, hor, ver, diag, rank, npoints/2);

      // Send boundaries to respective neighbours
      MPI_Send(row , npoints/2, MPI_DOUBLE, *ver , rank, MPI_COMM_WORLD);
      MPI_Send(col , npoints/2, MPI_DOUBLE, *hor , rank, MPI_COMM_WORLD);
      MPI_Send(indv, 1,         MPI_DOUBLE, *diag, rank, MPI_COMM_WORLD);

      // Compute turbulance: Receive neighbours
      MPI_Recv(row,  npoints/2, MPI_DOUBLE, *hor,  *hor,  MPI_COMM_WORLD, &status[rank]);
      MPI_Recv(col,  npoints/2, MPI_DOUBLE, *ver,  *ver,  MPI_COMM_WORLD, &status[rank]);
      MPI_Recv(indv, 1, MPI_DOUBLE, *diag, *diag, MPI_COMM_WORLD, &status[rank]);

      // Nine point stencil on CUDA cores
      evolve9ptCUDA<<<grid_dim, block_dim>>>(d_un, d_uc, d_uo, d_pebble, n, rank, row, col, indv, h, dt, t);
      d_temp = d_uc;
      d_uc = d_un;
      d_un = d_uo;
      d_uo = d_temp;

      if(!tpdt(&t,dt,end_time)) {
        break;
      }
    }

    hipMemcpy(un, d_un, sizeof(double) * n * n, hipMemcpyDeviceToHost);

    /*-----------------------------------*/
    // // Send final results to Rank 0.
    // MPI_Isend(un, size, MPI_DOUBLE, 0, rank+10,  MPI_COMM_WORLD, &request[0]);
    // printf("Done\n");
    /*-----------------------------------*/

    // Initial Output files
    char* s;
    s = (char*)malloc(sizeof(char)*17);
    int k = sprintf(s, "lake_node_%d.dat", rank);
    if (k>=0)
      print_heatmap(s, un, npoints/2, h);
    else {
      printf("Error in filename!\n");
      MPI_Finalize();
      return 0;
    }
  }

  MPI_Finalize();
  return 0;
}

void dest(double *source, double *row, double *col, double *indv, int *hor, int *ver, int *diag, int myrank, int size) {
  int i, x, y;
  switch (myrank) {
    case 1:
      for(i=0; i<size; i++) {
        x=size*(size-1)+i;
        y=i*size+(size-1);
        row[i]=source[x];
        col[i]=source[y];
      }
      *indv = source[(size-1)*(size-1)-1];
      *ver  = 2;
      *diag = 4;
      *hor  = 3;
      break;
    case 2:
      for(i=0; i<size; i++) {
        x=size*(size-1)+i;
        y=i;
        row[i]=source[x];
        col[i]=source[i];
      }
      *indv = source[(size-1)*(size-1)-1];
      *ver  = 1;
      *diag = 3;
      *hor  = 4;
      break;
    case 3:
      for(i=0; i<size; i++) {
        x=i;
        y=i*size+(size-1);
        row[i]=source[x];
        col[i]=source[y];
      }
      *indv = source[(size-1)*(size-1)-1];
      *ver  = 4;
      *diag = 2;
      *hor  = 1;
      break;
    case 4:
      for(i=0; i<size; i++) {
        x=i;
        y=i;
        row[i]=source[x];
        col[i]=source[y];
      }
      *indv = source[(size-1)*(size-1)-1];
      *ver  = 3;
      *diag = 1;
      *hor  = 2;
      break;
  }
}
void transfer(double *from, double *to, int r, int n, bool dir) {
  // This is really naive. I'll probably change it.
  int x,y, idx_t, idx_f;
  for (x=0; x<(int) n/2; x++)
  for (y=0; y<(int) n/2; y++) {
    if (r==1) {
      idx_t = x*n/2+y;
      idx_f = x*n+y;
    }
    else if (r==2) {
      idx_t = x*n/2+y;
      idx_f = x*n+n/2+y;
    }
    else if (r==3) {
      idx_t=x*n/2+y;
      idx_f=(x+n/2)*n+y;
    }
    else if (r==4) {
      idx_t=x*n/2+y;
      idx_f=(x+n/2)*n+y+n/2;
    }
    if (dir==true)
      to[idx_t]=from[idx_f];
    else
      to[idx_f]=from[idx_t];
  }
}

void init_pebbles(double *p, int pn, int n)
{
  int i, j, k, idx;
  int sz;

  srand( time(NULL) );
  memset(p, 0, sizeof(double) * n * n);

  for( k = 0; k < pn ; k++ )
  {
    i = rand() % (n - 4) + 2;
    j = rand() % (n - 4) + 2;
    sz = rand() % MAX_PSZ;
    idx = j + i * n;
    p[idx] = (double) sz;
  }
}

double f(double p, double t)
{
  return -expf(-TSCALE * t) * p;
}

int tpdt(double *t, double dt, double tf)
{
  if((*t) + dt > tf) return 0;
  (*t) = (*t) + dt;
  return 1;
}

void init(double *u, double *pebbles, int n)
{
  int i, j, idx;

  for(i = 0; i < n ; i++)
  {
    for(j = 0; j < n ; j++)
    {
      idx = j + i * n;
      u[idx] = f(pebbles[idx], 0.0);
    }
  }
}

void run_cpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time)
{
  double *un, *uc, *uo;
  double t, dt;

  un = (double*)malloc(sizeof(double) * n * n);
  uc = (double*)malloc(sizeof(double) * n * n);
  uo = (double*)malloc(sizeof(double) * n * n);

  memcpy(uo, u0, sizeof(double) * n * n);
  memcpy(uc, u1, sizeof(double) * n * n);

  t = 0.;
  dt = h / 2.;

  while(1)
  {

    evolve9pt_1(un, uc, uo, pebbles, n, h, dt, t);
    memcpy(uo, uc, sizeof(double) * n * n);
    memcpy(uc, un, sizeof(double) * n * n);

    if(!tpdt(&t,dt,end_time)) break;
  }

  memcpy(u, un, sizeof(double) * n * n);
}

void evolve9pt_1(double *un, double *uc, double *uo, double *pebbles, int n, double h, double dt, double t)
{
  int i, j, idx;

  for( i = 0; i < n; i++)
  {
    for( j = 0; j < n; j++)
    {
      idx = j + i * n;

      if( i == 0 || i == n - 1 || j == 0 || j == n - 1)
      {
        un[idx] = 0.;
      }
      else
      {
        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) *((uc[idx-1] + uc[idx+1] + uc[idx + n] + uc[idx - n] + 0.25*(uc[idx + n - 1] + uc[idx + n + 1] + uc[idx - n - 1] + uc[idx - n + 1])- 5 * uc[idx])/(h * h) + f(pebbles[idx],t));
      }
    }
  }
}


void print_heatmap(char *filename, double *u, int n, double h)
{
  int i, j, idx;

  FILE *fp = fopen(filename, "w");

  for( i = 0; i < n; i++ )
  {
    for( j = 0; j < n; j++ )
    {
      idx = j + i * n;
      fprintf(fp, "%f %f %0.2e\n", i*h, j*h, u[idx]);
    }
  }

  fclose(fp);
}
